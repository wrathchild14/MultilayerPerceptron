#include "hip/hip_runtime.h"
#include "mlp.h"

__device__ double atomicAddDouble(double *address, double val)
{
	auto address_as_ull = (unsigned long long *)address;
	unsigned long long old_val = *address_as_ull, new_val;
	do
	{
		new_val = __double_as_longlong(__longlong_as_double(old_val) + val);
	} while (atomicCAS(address_as_ull, old_val, new_val) != old_val);
	return __longlong_as_double(old_val);
}

__device__ double activation(const double x)
{
	return tanh(x);
}

__device__ double activation_derivative(const double x)
{
	return 1 - pow(tanh(x), 2);
}

__global__ void forward_kernel(const int input_size, const int hidden_size, const int output_size,
							   const double *input, const double *w1, const double *w2,
							   const double *b1, const double *b2, double *hidden, double *output)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < hidden_size)
	{
		double sum = 0.0;
		for (int j = 0; j < input_size; j++)
		{
			sum += input[j] * w1[j * hidden_size + i];
		}
		hidden[i] = activation(sum + b1[i]);
	}

	if (i < output_size)
	{
		double sum = 0.0;
		for (int j = 0; j < hidden_size; j++)
		{
			sum += hidden[j] * w2[j * output_size + i];
		}
		output[i] = activation(sum + b2[i]);
	}
}

__global__ void backward_kernel(const int input_size, const int hidden_size, const int output_size,
								const double *input, const double *target, const double *w1, const double *w2,
								const double *b1, const double *b2, double *hidden, double *output,
								double *w1_gradient, double *w2_gradient, double *b1_gradient, double *b2_gradient)
{
	unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

	// output error terms
	if (i < output_size)
	{
		const double output1 = output[i];
		const double output_error = (output1 - target[i]) * activation_derivative(output1);
		for (int j = 0; j < hidden_size; j++)
		{
			atomicAddDouble(&w2_gradient[j * output_size + i], output_error * hidden[j]);
		}
		atomicAddDouble(&b2_gradient[i], output_error);
	}

	// hidden error terms
	if (i < hidden_size)
	{
		double error = 0.0;
		for (int j = 0; j < output_size; j++)
		{
			error += (output[j] - target[j]) * w2[i * output_size + j];
		}
		const double hidden_error = error * activation_derivative(hidden[i]);
		for (int j = 0; j < input_size; j++)
		{
			atomicAddDouble(&w1_gradient[j * hidden_size + i], hidden_error * input[j]);
		}
		atomicAddDouble(&b1_gradient[i], hidden_error);
	}
}

void train(mlp *network, double **inputs, double **labels, int num_samples, double learning_rate,
		   int epochs, int batch_size)
{
	double *d_w1, *d_w2, *d_w1_gradient, *d_w2_gradient;
	double *d_b1, *d_b2, *d_hidden, *d_output;

	hipMalloc((void **)&d_w1, network->input_size * network->hidden_size * sizeof(double));
	hipMalloc((void **)&d_w2, network->hidden_size * network->output_size * sizeof(double));
	hipMalloc((void **)&d_w1_gradient, network->input_size * network->hidden_size * sizeof(double));
	hipMalloc((void **)&d_w2_gradient, network->hidden_size * network->output_size * sizeof(double));
	hipMalloc((void **)&d_b1, network->hidden_size * sizeof(double));
	hipMalloc((void **)&d_b2, network->output_size * sizeof(double));
	hipMalloc((void **)&d_hidden, network->hidden_size * sizeof(double));
	hipMalloc((void **)&d_output, network->output_size * sizeof(double));

	// host to gpu
	hipMemcpy(d_w1, network->w1[0], network->input_size * network->hidden_size * sizeof(double),
			   hipMemcpyHostToDevice);
	hipMemcpy(d_w2, network->w2[0], network->hidden_size * network->output_size * sizeof(double),
			   hipMemcpyHostToDevice);
	hipMemcpy(d_b1, network->b1, network->hidden_size * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_b2, network->b2, network->output_size * sizeof(double), hipMemcpyHostToDevice);

	for (int epoch = 0; epoch < epochs; epoch++)
	{
		for (int batch = 0; batch < num_samples; batch += batch_size)
		{
			int end = batch + batch_size;
			if (end > num_samples)
				end = num_samples;

			double *d_w1_gradient, *d_w2_gradient;
			double *d_b1_gradient, *d_b2_gradient;

			hipMalloc((void **)&d_w1_gradient, network->input_size * network->hidden_size * sizeof(double));
			hipMalloc((void **)&d_w2_gradient, network->hidden_size * network->output_size * sizeof(double));
			hipMalloc((void **)&d_b1_gradient, network->hidden_size * sizeof(double));
			hipMalloc((void **)&d_b2_gradient, network->output_size * sizeof(double));

			// init gradients to zero
			hipMemset(d_w1_gradient, 0, network->input_size * network->hidden_size * sizeof(double));
			hipMemset(d_w2_gradient, 0, network->hidden_size * network->output_size * sizeof(double));
			hipMemset(d_b1_gradient, 0, network->hidden_size * sizeof(double));
			hipMemset(d_b2_gradient, 0, network->output_size * sizeof(double));

			// weights to gpu
			hipMemcpy(d_w1_gradient, network->w1, network->input_size * sizeof(double *), hipMemcpyHostToDevice);
			hipMemcpy(d_w2_gradient, network->w2, network->hidden_size * sizeof(double *), hipMemcpyHostToDevice);

			for (int sample = batch; sample < end; sample++)
			{
				const double *input = inputs[sample];
				const double *target = labels[sample];

				forward_kernel<<<network->hidden_size / 256 + 1, 256>>>(network->input_size, network->hidden_size,
																		network->output_size, input, d_w1, d_w2,
																		d_b1, d_b2, d_hidden, d_output);
				hipDeviceSynchronize();

				backward_kernel<<<network->hidden_size / 256 + 1, 256>>>(network->input_size, network->hidden_size,
																		 network->output_size, input, target, d_w1,
																		 d_w2, d_b1, d_b2, d_hidden, d_output,
																		 d_w1_gradient, d_w2_gradient,
																		 d_b1_gradient, d_b2_gradient);
				hipDeviceSynchronize();
			}

			const double factor = learning_rate / static_cast<double>(batch_size);
			for (int i = 0; i < network->input_size; i++)
			{
				hipMemcpy(network->w1[i], d_w1, network->hidden_size * sizeof(double), hipMemcpyDeviceToHost);
				for (int j = 0; j < network->hidden_size; j++)
				{
					network->w1[i][j] -= factor * d_w1_gradient[i * network->hidden_size + j];
				}
			}
			for (int i = 0; i < network->hidden_size; i++)
			{
				hipMemcpy(network->w2[i], d_w2, network->output_size * sizeof(double), hipMemcpyDeviceToHost);
				for (int j = 0; j < network->output_size; j++)
				{
					network->w2[i][j] -= factor * d_w2_gradient[i * network->output_size + j];
				}
			}
			hipMemcpy(network->b1, d_b1, network->hidden_size * sizeof(double), hipMemcpyDeviceToHost);
			hipMemcpy(network->b2, d_b2, network->output_size * sizeof(double), hipMemcpyDeviceToHost);

			hipFree(d_w1_gradient);
			hipFree(d_w2_gradient);
			hipFree(d_b1_gradient);
			hipFree(d_b2_gradient);
		}

		printf("epoch %d/%d, loss %lf\n", epoch + 1, epochs, network->loss);
	}

	hipMemcpy(network->w1, d_w1, network->input_size * network->hidden_size * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(network->w2, d_w2, network->hidden_size * network->output_size * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(network->b1, d_b1, network->hidden_size * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(network->b2, d_b2, network->output_size * sizeof(double), hipMemcpyDeviceToHost);

	hipFree(d_w1);
	hipFree(d_w2);
	hipFree(d_b1);
	hipFree(d_b2);
	hipFree(d_hidden);
	hipFree(d_output);

	// printf("training done in %f s\n", omp_get_wtime() - epochs_dt);
}

mlp *create_mlp(int input_size, int hidden_size, int output_size)
{
	mlp *network = static_cast<mlp *>(malloc(sizeof(mlp)));
	network->input_size = input_size;
	network->hidden_size = hidden_size;
	network->output_size = output_size;

	network->w1 = static_cast<double **>(malloc(input_size * sizeof(double *)));
	network->w2 = static_cast<double **>(malloc(hidden_size * sizeof(double *)));
	network->b1 = static_cast<double *>(malloc(hidden_size * sizeof(double)));
	network->b2 = static_cast<double *>(malloc(output_size * sizeof(double)));

	network->hidden = static_cast<double *>(malloc(hidden_size * sizeof(double)));
	network->output = static_cast<double *>(malloc(output_size * sizeof(double)));

	for (int i = 0; i < input_size; i++)
	{
		network->w1[i] = static_cast<double *>(malloc(hidden_size * sizeof(double)));
	}

	for (int i = 0; i < hidden_size; i++)
	{
		network->w2[i] = static_cast<double *>(malloc(output_size * sizeof(double)));
	}

	for (int i = 0; i < input_size; i++)
	{
		for (int j = 0; j < hidden_size; j++)
		{
			network->w1[i][j] = ((double)rand() / RAND_MAX) * 2 - 1;
		}
	}

	for (int i = 0; i < hidden_size; i++)
	{
		for (int j = 0; j < output_size; j++)
		{
			network->w2[i][j] = ((double)rand() / RAND_MAX) * 2 - 1;
		}
	}

	for (int i = 0; i < hidden_size; i++)
	{
		network->b1[i] = 0;
	}

	for (int i = 0; i < output_size; i++)
	{
		network->b2[i] = 0;
	}

	return network;
}

void free_network(mlp *network)
{
	for (int i = 0; i < network->input_size; i++)
	{
		free(network->w1[i]);
	}
	free(network->w1);

	for (int i = 0; i < network->hidden_size; i++)
	{
		free(network->w2[i]);
	}
	free(network->w2);

	free(network->b1);
	free(network->b2);
	free(network->hidden);
	free(network->output);

	free(network);
}
